#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>

const int M = 2;  // Rows of A
const int N = 3;  // Columns of A and Rows of B
const int P = 4;  // Columns of B
const int K = 3;  // Number of matrices

__global__ void matrixMultiplication(int *A, int *B, int *C, int m, int n, int p) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < p) {
        int sum = 0;
        for (int k = 0; k < n; ++k) {
            sum += A[row * n + k] * B[k * p + col];
        }
        C[row * p + col] = sum;
    }
}

void printMatrix(int *matrix, int rows, int cols, const char *name) {
    std::cout << name << ":\n";
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            std::cout << matrix[i * cols + j] << "\t";
        }
        std::cout << "\n";
    }
    std::cout << "\n";
}

void initializeRandomMatrix(int *matrix, int rows, int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            matrix[i * cols + j] = rand() % 10;  // Adjust the range as needed
        }
    }
}

int main() {
    srand(time(NULL));  // Seed for random number generation

    int *h_A, *h_B, *h_C;  // Host matrices
    int *d_A, *d_B, *d_C;  // Device matrices

    // Allocate host memory
    h_A = (int *)malloc(K * M * N * sizeof(int));
    h_B = (int *)malloc(K * N * P * sizeof(int));
    h_C = (int *)malloc(K * M * P * sizeof(int));

    // Initialize host matrices A and B with random data
    for (int k = 0; k < K; ++k) {
        initializeRandomMatrix(h_A + k * M * N, M, N);
        initializeRandomMatrix(h_B + k * N * P, N, P);
    }

    // Print input matrices
    for (int k = 0; k < K; ++k) {
        printMatrix(h_A + k * M * N, M, N, ("Matrix A" + std::to_string(k)).c_str());
        printMatrix(h_B + k * N * P, N, P, ("Matrix B" + std::to_string(k)).c_str());
    }

    // Allocate device memory
    hipMalloc((void **)&d_A, K * M * N * sizeof(int));
    hipMalloc((void **)&d_B, K * N * P * sizeof(int));
    hipMalloc((void **)&d_C, K * M * P * sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_A, h_A, K * M * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, K * N * P * sizeof(int), hipMemcpyHostToDevice);

    // Define thread block and grid dimensions
    dim3 blockDim(16, 16);
    dim3 gridDim((P + blockDim.x - 1) / blockDim.x, (M + blockDim.y - 1) / blockDim.y);

    // Launch the matrix multiplication kernel for each pair of matrices
    for (int k = 0; k < K; ++k) {
        matrixMultiplication<<<gridDim, blockDim>>>(d_A + k * M * N, d_B + k * N * P, d_C + k * M * P, M, N, P);
    }

    // Copy the result back to the host
    hipMemcpy(h_C, d_C, K * M * P * sizeof(int), hipMemcpyDeviceToHost);

    // Print output matrices
    for (int k = 0; k < K; ++k) {
        printMatrix(h_C + k * M * P, M, P, ("Result Matrix" + std::to_string(k)).c_str());
    }

    // Free device and host memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}